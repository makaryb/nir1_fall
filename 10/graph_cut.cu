#include "hip/hip_runtime.h"
/********************************************************************/
/***** GPU Graph Cut ************************************************/
/********************************************************************/
////////////////////////////////////////////////////
// Copyright (c) 2018 Kiyoshi Oguri    2018.02.14 //
// Released under the MIT license                 //
// http://opensource.org/licenses/mit-license.php //
////////////////////////////////////////////////////

extern int cost(int S, int H, int W);
extern void cut(int S, int H, int W);

#define TD_NUM (256)
#define LOOP (37)

int SIZE_S;
int SIZE_H;
int SIZE_W;
int SIZE_HW;
int SIZE_SHW;

__constant__ int size_s;
__constant__ int size_h;
__constant__ int size_w;
__constant__ int size_hw;
__constant__ int size_shw;

int *h_FLG;
int *h_FLW;
int *h_OVF;
int *h_HGT;

int *d_FLG;
int *d_FLW;
int *d_OVF;
int *d_HGT;
int *d_TAG;

__constant__ int *FLW;
__constant__ int *OVF;
__constant__ int *HGT;
__constant__ int *TAG;

size_t I_SIZE;
size_t FLW_SIZE;
size_t OVF_SIZE;
size_t HGT_SIZE;
size_t TAG_SIZE;

#define h_ADR1(S, H, W) \
 ((S)*SIZE_HW+\
  (H)*SIZE_W+\
  (W))

#define h_ADR2(S, H, W, D) \
 ((D)*SIZE_SHW+\
  (S)*SIZE_HW+\
  (H)*SIZE_W+\
  (W))

#define ADR1(S, H, W) \
 ((S)*size_hw+\
  (H)*size_w+\
  (W))

#define ADR2(S, H, W, D) \
 ((D)*size_shw+\
  (S)*size_hw+\
  (H)*size_w+\
  (W))

#define SADR1(S, H, W) \
 ((S)*SIZE_BHW+\
  (H)*SIZE_BW+\
  (W))

#define DIV(A,B) (((A)%(B)==0)? ((A)/(B)) : (((A)/(B))+1))
#define SIZE_BS (8)
#define SIZE_BH (8)
#define SIZE_BW (8)
#define SIZE_BSHW (SIZE_BS*SIZE_BH*SIZE_BW)
#define SIZE_BHW (SIZE_BH*SIZE_BW)
#define SIZE_GS DIV(SIZE_S,SIZE_BS)
#define SIZE_GH DIV(SIZE_H,SIZE_BH)
#define SIZE_GW DIV(SIZE_W,SIZE_BW)

#define Out_Mask 0x000003ff

#define Out0_Set 0x00000001
#define Out1_Set 0x00000002
#define Out2_Set 0x00000004
#define Out3_Set 0x00000008
#define Out4_Set 0x00000010
#define Out5_Set 0x00000020
#define Out6_Set 0x00000040
#define Out7_Set 0x00000080
#define Out8_Set 0x00000100
#define Out9_Set 0x00000200

#define Out0_Rst ~Out0_Set
#define Out1_Rst ~Out1_Set
#define Out2_Rst ~Out2_Set
#define Out3_Rst ~Out3_Set
#define Out4_Rst ~Out4_Set
#define Out5_Rst ~Out5_Set
#define Out6_Rst ~Out6_Set
#define Out7_Rst ~Out7_Set
#define Out8_Rst ~Out8_Set
#define Out9_Rst ~Out9_Set

inline __device__ int edg_read(int S, int H, int W, int D) {
  return FLW[ADR2(S, H, W, D)];
}

inline __device__ void edg_add(int S, int H, int W, int D, int V) {
  FLW[ADR2(S, H, W, D)] += V;
}

inline __device__ int ovf_read(int S, int H, int W) {
  return OVF[ADR1(S, H, W)];
}

inline __device__ void ovf_add(int S, int H, int W, int V) {
  atomicAdd(&(OVF[ADR1(S, H, W)]), V);
}

inline __device__ int hgt_read(int S, int H, int W) {
  return HGT[ADR1(S, H, W)];
}

inline __device__ void hgt_write(int S, int H, int W, int V) {
  HGT[ADR1(S, H, W)] = V;
}

inline __device__ int tag_read(int S, int H, int W) {
  return TAG[ADR1(S, H, W)];
}

inline __device__ void tag_set(int S, int H, int W, int M) {
  atomicOr(&(TAG[ADR1(S, H, W)]), M);
}

inline __device__ void tag_rst(int S, int H, int W, int M) {
  atomicAnd(&(TAG[ADR1(S, H, W)]), M);
}

inline __device__ int shed_read(int SM[], int S, int H, int W) {
  return SM[SADR1(S, H, W)];
}

inline __device__ void shed_write(int SM[], int S, int H, int W, int V) {
  SM[SADR1(S, H, W)] = V;
}

__global__ void reset(int FLG[]) {
  FLG[0] = 0;
}

__device__ void out_chk(int S, int H, int W, int D, int SM) {
  if (edg_read(S, H, W, D) > 0) tag_set(S, H, W, SM);
}

__global__ void tag_init(void) {
  ///////////////////////////////
  int total_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (total_id >= size_shw) return;
  int S  = total_id / size_hw;
  int sa = total_id % size_hw;
  int H  = sa       / size_w;
  int W  = sa       % size_w;
  ///////////////////////////////
  tag_rst(S, H, W, ~Out_Mask);
  out_chk(S, H, W, 0, Out0_Set);
  out_chk(S, H, W, 1, Out1_Set);
  out_chk(S, H, W, 2, Out2_Set);
  out_chk(S, H, W, 3, Out3_Set);
  out_chk(S, H, W, 4, Out4_Set);
  out_chk(S, H, W, 5, Out5_Set);
  out_chk(S, H, W, 6, Out6_Set);
  out_chk(S, H, W, 7, Out7_Set);
  out_chk(S, H, W, 8, Out8_Set);
  out_chk(S, H, W, 9, Out9_Set);
}

__global__ void bfs_init(void) {
  ///////////////////////////////
  int total_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (total_id >= size_shw) return;
  int S  = total_id / size_hw;
  int sa = total_id % size_hw;
  int H  = sa       / size_w;
  int W  = sa       % size_w;
  ///////////////////////////////
  if (ovf_read(S, H, W) < 0) hgt_write(S, H, W, 0);
  else                       hgt_write(S, H, W, size_shw);
}

__device__ void bfs_front_i(int SM[], int S, int H, int W, int s, int h, int w, int &nt) {
  ///////////////////////////////
  int nS = S + s;
  int nH = H + h;
  int nW = W + w;
  ///////////////////////////////
  int T = shed_read(SM, nS, nH, nW) + 1;
  if (nt > T) nt = T;
}

__global__ void bfs_do_i(void) {
  __shared__ int FLG;
  __shared__ int SM[SIZE_BSHW];
  ///////////////////////////////
  int bs = blockIdx.z;
  int bh = blockIdx.y;
  int bw = blockIdx.x;
  //---------------------------//
  int s = threadIdx.z;
  int h = threadIdx.y;
  int w = threadIdx.x;
  //---------------------------//
  int S = SIZE_BS * bs + s;
  if (S >= size_s) return;
  int H = SIZE_BH * bh + h;
  if (H >= size_h) return;
  int W = SIZE_BW * bw + w;
  if (W >= size_w) return;
  ///////////////////////////////
  bool nzz = (s != 0);
  bool nzy = (h != 0);
  bool nzx = (w != 0);
  bool niz = ((s != (SIZE_BS-1)) && (S != (size_s-1)));
  bool niy = ((h != (SIZE_BH-1)) && (H != (size_h-1)));
  bool nix = ((w != (SIZE_BW-1)) && (W != (size_w-1)));
  ///////////////////////////////
  int tag = tag_read(S, H, W);
  shed_write(SM, s, h, w, hgt_read(S, H, W));
  for ( ; ; ) {
    __syncthreads();
    if ((s == 0) && (h == 0) && (w == 0)) FLG = 0;
    __syncthreads();
    if (tag & Out_Mask) {
      int ct = shed_read(SM, s, h, w);
      int nt = ct;
      if (niz       ) if (tag & Out0_Set) bfs_front_i(SM, s, h, w, 1, 0, 0, nt);
      if (niz && nix) if (tag & Out5_Set) bfs_front_i(SM, s, h, w, 1, 0, 1, nt);
      if (niz && nzx) if (tag & Out6_Set) bfs_front_i(SM, s, h, w, 1, 0,-1, nt);
      if (       niy) if (tag & Out2_Set) bfs_front_i(SM, s, h, w, 0, 1, 0, nt);
      if (       nzy) if (tag & Out1_Set) bfs_front_i(SM, s, h, w, 0,-1, 0, nt);
      if (       nix) if (tag & Out4_Set) bfs_front_i(SM, s, h, w, 0, 0, 1, nt);
      if (       nzx) if (tag & Out3_Set) bfs_front_i(SM, s, h, w, 0, 0,-1, nt);
      if (nzz && nix) if (tag & Out8_Set) bfs_front_i(SM, s, h, w,-1, 0, 1, nt);
      if (nzz && nzx) if (tag & Out7_Set) bfs_front_i(SM, s, h, w,-1, 0,-1, nt);
      if (nzz       ) if (tag & Out9_Set) bfs_front_i(SM, s, h, w,-1, 0, 0, nt);
      if (nt != ct) {
        shed_write(SM, s, h, w, nt);
        if (FLG == 0) FLG = 1;
      }
    }
    __syncthreads();
    if (FLG == 0) break;
  }
  hgt_write(S, H, W, shed_read(SM, s, h, w));
}

__device__ void bfs_front_o(int S, int H, int W, int s, int h, int w, int &nt) {
  ///////////////////////////////
  int nS = S + s;
  int nH = H + h;
  int nW = W + w;
  ///////////////////////////////
  int T = hgt_read(nS, nH, nW) + 1;
  if (nt > T) nt = T;
}

__global__ void bfs_do_o(int FLG[]) {
  ///////////////////////////////
  int bs = blockIdx.z;
  int bh = blockIdx.y;
  int bw = blockIdx.x;
  //---------------------------//
  int s = threadIdx.z;
  int h = threadIdx.y;
  int w = threadIdx.x;
  //---------------------------//
  int S = SIZE_BS * bs + s;
  if (S >= size_s) return;
  int H = SIZE_BH * bh + h;
  if (H >= size_h) return;
  int W = SIZE_BW * bw + w;
  if (W >= size_w) return;
  ///////////////////////////////
  bool pzz = ((s == 0) && (S != 0));
  bool pzy = ((h == 0) && (H != 0));
  bool pzx = ((w == 0) && (W != 0));
  bool piz = ((s == (SIZE_BS-1)) && (S != (size_s-1)));
  bool piy = ((h == (SIZE_BH-1)) && (H != (size_h-1)));
  bool pix = ((w == (SIZE_BW-1)) && (W != (size_w-1)));
  ///////////////////////////////
  int tag = tag_read(S, H, W);
  if (tag & Out_Mask) {
    int ct = hgt_read(S, H, W);
    int nt = ct;
    if (piz       ) if (tag & Out0_Set) bfs_front_o(S, H, W, 1, 0, 0, nt);
    if (piz || pix) if (tag & Out5_Set) bfs_front_o(S, H, W, 1, 0, 1, nt);
    if (piz || pzx) if (tag & Out6_Set) bfs_front_o(S, H, W, 1, 0,-1, nt);
    if (       piy) if (tag & Out2_Set) bfs_front_o(S, H, W, 0, 1, 0, nt);
    if (       pzy) if (tag & Out1_Set) bfs_front_o(S, H, W, 0,-1, 0, nt);
    if (       pix) if (tag & Out4_Set) bfs_front_o(S, H, W, 0, 0, 1, nt);
    if (       pzx) if (tag & Out3_Set) bfs_front_o(S, H, W, 0, 0,-1, nt);
    if (pzz || pix) if (tag & Out8_Set) bfs_front_o(S, H, W,-1, 0, 1, nt);
    if (pzz || pzx) if (tag & Out7_Set) bfs_front_o(S, H, W,-1, 0,-1, nt);
    if (pzz       ) if (tag & Out9_Set) bfs_front_o(S, H, W,-1, 0, 0, nt);
    if (nt != ct) {
      hgt_write(S, H, W, nt);
      if (FLG[0] == 0) FLG[0] = 1;
    }
  }
}

__global__ void ovf_do(int FLG[]) {
  if (FLG[0] != 0) return;
  ///////////////////////////////
  int total_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (total_id >= size_shw) return;
  int S  = total_id / size_hw;
  int sa = total_id % size_hw;
  int H  = sa       / size_w;
  int W  = sa       % size_w;
  ///////////////////////////////
  if (hgt_read(S, H, W) == size_shw) return;
  if (ovf_read(S, H, W) <= 0) return;
  FLG[0] = 1;
}

__device__ void push1(int S, int H, int W, int s, int h, int w, int D, int R, int SM, int RM, int hh, int &oo) {
  ///////////////////////////////
  int nS = S + s;
  int nH = H + h;
  int nW = W + w;
  ///////////////////////////////
  if (hgt_read(nS, nH, nW) >= hh) return;
  int mm = edg_read(S, H, W, D);
  bool qq = oo >= mm;
  int pp = qq? mm: oo;
  ovf_add(nS, nH, nW, pp);
  ovf_add(S, H, W, -pp);
  edg_add(nS, nH, nW, R, pp);
  edg_add(S, H, W, D, -pp);
  oo -= pp;
  if (qq) tag_rst(S, H, W, RM);
  tag_set(nS, nH, nW, SM);
}

__global__ void push(void) {
  ///////////////////////////////
  int total_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (total_id >= size_shw) return;
  int S  = total_id / size_hw;
  int sa = total_id % size_hw;
  int H  = sa       / size_w;
  int W  = sa       % size_w;
  ///////////////////////////////
  int hh = hgt_read(S, H, W);
  if (hh == size_shw) return;
  int oo = ovf_read(S, H, W);
  if (oo <= 0) return;
  int tag = tag_read(S, H, W);
  if ((tag & Out_Mask) == 0) return;
  if (tag & Out0_Set) push1(S, H, W, 1, 0, 0, 0, 9, Out9_Set, Out0_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out5_Set) push1(S, H, W, 1, 0, 1, 5, 7, Out7_Set, Out5_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out6_Set) push1(S, H, W, 1, 0,-1, 6, 8, Out8_Set, Out6_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out2_Set) push1(S, H, W, 0, 1, 0, 2, 1, Out1_Set, Out2_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out1_Set) push1(S, H, W, 0,-1, 0, 1, 2, Out2_Set, Out1_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out4_Set) push1(S, H, W, 0, 0, 1, 4, 3, Out3_Set, Out4_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out3_Set) push1(S, H, W, 0, 0,-1, 3, 4, Out4_Set, Out3_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out8_Set) push1(S, H, W,-1, 0, 1, 8, 6, Out6_Set, Out8_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out7_Set) push1(S, H, W,-1, 0,-1, 7, 5, Out5_Set, Out7_Rst, hh, oo); if (oo <= 0) return;
  if (tag & Out9_Set) push1(S, H, W,-1, 0, 0, 9, 0, Out0_Set, Out9_Rst, hh, oo); if (oo <= 0) return;
  hgt_write(S, H, W, hh + 1);
}

void push_relabel(int loop) {
  dim3 grid(SIZE_GW, SIZE_GH, SIZE_GS);
  dim3 block(SIZE_BW, SIZE_BH, SIZE_BS);
  tag_init<<< (SIZE_SHW/TD_NUM)+1, TD_NUM >>>();
  for ( ; ; ) {
    bfs_init<<< (SIZE_SHW/TD_NUM)+1, TD_NUM >>>();
    for ( ; ; ) {
      bfs_do_i<<< grid, block >>>();
      reset<<< 1, 1 >>>(d_FLG);
      bfs_do_o<<< grid, block >>>(d_FLG);
      hipDeviceSynchronize();
      hipMemcpy(h_FLG, d_FLG, I_SIZE, hipMemcpyDeviceToHost);
      if (h_FLG[0] == 0) break;
    }
    reset<<< 1, 1 >>>(d_FLG);
    ovf_do<<< (SIZE_SHW/TD_NUM)+1, TD_NUM >>>(d_FLG);
    hipDeviceSynchronize();
    hipMemcpy(h_FLG, d_FLG, I_SIZE, hipMemcpyDeviceToHost);
    if (h_FLG[0] == 0) break;
    for (int i = 0; i < loop; i++) {
      push<<< (SIZE_SHW/TD_NUM)+1, TD_NUM >>>();
    }
  }
}

void data_set(int penalty_w, int penalty_h, int inhibit_a, int inhibit_b) {
  for (int H = 0; H < SIZE_H; H++) {
    for (int W = 0; W < SIZE_W; W++) {
      for (int S = 0; S < SIZE_S; S++) {
        ///////////////////////////////
        for (int i = 0; i < 10; i++) h_FLW[h_ADR2(S, H, W, i)] = 0;
        h_OVF[h_ADR1(S, H, W)] = 0;
        ///////////////////////////////
        if  (S!=SIZE_S-1)          h_FLW[h_ADR2(S, H, W, 0)] = cost(S+1, H, W);
        if  (S==SIZE_S-1)          h_OVF[h_ADR1(S, H, W)]   -= cost(S+1, H, W);
        if  (S==0)                 h_OVF[h_ADR1(S, H, W)]   += cost(S,   H, W);
        if  (S!=0)                 h_FLW[h_ADR2(S, H, W, 9)] = inhibit_a;
        if          (H!=0)         h_FLW[h_ADR2(S, H, W, 1)] = penalty_h;
        if          (H!=SIZE_H-1)  h_FLW[h_ADR2(S, H, W, 2)] = penalty_h;
        if          (W!=0)         h_FLW[h_ADR2(S, H, W, 3)] = penalty_w;
        if          (W!=SIZE_W-1)  h_FLW[h_ADR2(S, H, W, 4)] = penalty_w;
        if ((S!=0)&&(W!=0))        h_FLW[h_ADR2(S, H, W, 7)] = inhibit_b;
        if ((S!=0)&&(W!=SIZE_W-1)) h_FLW[h_ADR2(S, H, W, 8)] = inhibit_b;
      }
    }
  }
  hipMemcpy(d_FLW, h_FLW, FLW_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_OVF, h_OVF, OVF_SIZE, hipMemcpyHostToDevice);
}

int sink_chk(void) {
  hipMemcpy(h_OVF, d_OVF, OVF_SIZE, hipMemcpyDeviceToHost);
  int total = 0;
  for (int H = 0; H < SIZE_H; H++) {
    for (int W = 0; W < SIZE_W; W++) {
      int sink = h_OVF[h_ADR1(SIZE_S-1, H, W)];
      if (sink < 0) total += -sink;
    }
  }
  return total;
}

void dep_set(void) {
  hipMemcpy(h_HGT, d_HGT, HGT_SIZE, hipMemcpyDeviceToHost);
  for (int H = 0; H < SIZE_H; H++) {
    for (int W = 0; W < SIZE_W; W++) {
      for (int S = SIZE_S; S >= 0; S--) {
        if (S == SIZE_S) {
          if (h_HGT[h_ADR1(S-1, H, W)] == SIZE_SHW) {
            cut(S, H, W);
            break;
          }
        }
        else if (S == 0) {
          if (h_HGT[h_ADR1(S, H, W)] != SIZE_SHW) {
            cut(S, H, W);
            break;
          }
        }
        else {
          if ((h_HGT[h_ADR1(S, H, W)] != SIZE_SHW) && (h_HGT[h_ADR1(S-1, H, W)] == SIZE_SHW)) {
            cut(S, H, W);
            break;
          }
        }
      }
    }
  }
}

int graph_cut(int penalty_w, int penalty_h, int inhibit_a, int inhibit_b) {
  hipMemcpyToSymbol(HIP_SYMBOL(size_s),   &SIZE_S,   sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(size_h),   &SIZE_H,   sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(size_w),   &SIZE_W,   sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(size_hw),  &SIZE_HW,  sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(size_shw), &SIZE_SHW, sizeof(int));
  I_SIZE = sizeof(int);
  FLW_SIZE = sizeof(int) * SIZE_SHW*10;
  OVF_SIZE = sizeof(int) * SIZE_SHW;
  HGT_SIZE = sizeof(int) * SIZE_SHW;
  TAG_SIZE = sizeof(int) * SIZE_SHW;
  h_FLG = new int[1];
  h_FLW = new int[SIZE_SHW*10];
  h_OVF = new int[SIZE_SHW];
  h_HGT = new int[SIZE_SHW];
  hipMalloc((void **)&d_FLG, I_SIZE);
  hipMalloc((void **)&d_FLW, FLW_SIZE);
  hipMalloc((void **)&d_OVF, OVF_SIZE);
  hipMalloc((void **)&d_HGT, HGT_SIZE);
  hipMalloc((void **)&d_TAG, TAG_SIZE);
  hipMemcpyToSymbol(HIP_SYMBOL(FLW), &d_FLW, sizeof(int*));
  hipMemcpyToSymbol(HIP_SYMBOL(OVF), &d_OVF, sizeof(int*));
  hipMemcpyToSymbol(HIP_SYMBOL(HGT), &d_HGT, sizeof(int*));
  hipMemcpyToSymbol(HIP_SYMBOL(TAG), &d_TAG, sizeof(int*));
  data_set(penalty_w, penalty_h, inhibit_a, inhibit_b);
  int before = sink_chk();
  push_relabel(LOOP);
  int after = sink_chk();
  dep_set();
  hipFree(d_TAG);
  hipFree(d_HGT);
  hipFree(d_OVF);
  hipFree(d_FLW);
  hipFree(d_FLG);
  delete [] h_HGT;
  delete [] h_OVF;
  delete [] h_FLW;
  delete [] h_FLG;
  return before - after;
}
