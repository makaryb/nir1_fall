#include "hip/hip_runtime.h"
/********************************************************************/
/**************************************** 2018/01/30 *** K. Oguri ***/
/********************************************************************/
#include <stdio.h>
#include <math.h>
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <GL/freeglut.h>
#include <opencv2/highgui/highgui.hpp>

#define XX 640
#define YY 480
#define PI (3.141592653589793238462643383279502884197f)
#define LOOK_AT_R (4.0f)
#define LOOK_AT_1 (PI/6)
#define LOOK_AT_2 (PI/6)

#define TD_NUM   (256)
#define FRM_NUM (YY*XX*3)

#define PADR(Y, X, C) \
 ((Y)*3*XX+\
  (X)*3+\
  (C))

float look_at_r;
float look_at_1;
float look_at_2;

struct vvv {
  float x;
  float y;
  float z;
  float r;
  float g;
  float b;
};
hipGraphicsResource *v_res;

int *h_FRM;
int *d_FRM;
__constant__ int *FRM;
size_t FRM_SIZE = sizeof(int) * FRM_NUM;

cv::Mat frame(YY, XX, CV_8UC3, cv::Scalar(0, 0, 0));
cv::VideoCapture *cap;

/********************************************************************/
void display(void) {
  float p = look_at_r * cos(look_at_1);
  float z = look_at_r * sin(look_at_1);
  float x =         p * cos(look_at_2);
  float y =         p * sin(look_at_2);
  glLoadIdentity();
  if (p >= 0.0) gluLookAt(x, y, z, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0);
  else          gluLookAt(x, y, z, 0.0, 0.0, 0.0, 0.0, 0.0,-1.0);
  /*---------------------------*/
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glEnable(GL_DEPTH_TEST);
  /*---------------------------*/
  glEnableClientState(GL_VERTEX_ARRAY);
  glEnableClientState(GL_COLOR_ARRAY);
  glVertexPointer(3, GL_FLOAT, sizeof(vvv), NULL);
  glColorPointer(3, GL_FLOAT, sizeof(vvv), (GLvoid *)(sizeof(float)*3));
  glPointSize(1);
  glDrawArrays(GL_POINTS, 0, XX*YY);
  glDisableClientState(GL_COLOR_ARRAY);
  glDisableClientState(GL_VERTEX_ARRAY);
  /*---------------------------*/
  glColor3f(1.0, 0.0, 0.0); glBegin(GL_LINES); glVertex3f(-1.0, 0.0, 0.0); glVertex3f(1.0, 0.0, 0.0); glEnd();
  glColor3f(0.0, 1.0, 0.0); glBegin(GL_LINES); glVertex3f( 0.0,-1.0, 0.0); glVertex3f(0.0, 1.0, 0.0); glEnd();
  glColor3f(0.0, 0.0, 1.0); glBegin(GL_LINES); glVertex3f( 0.0, 0.0,-1.0); glVertex3f(0.0, 0.0, 1.0); glEnd();
  /*---------------------------*/
  glDisable(GL_DEPTH_TEST);
  glutSwapBuffers();
}

void reshape(int w, int h) {
  glViewport(0, 0, w, h);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluPerspective(30, (float)w / h, 0.001, 1000.0);
  glMatrixMode(GL_MODELVIEW);
}

void keyboard(unsigned char key, int x, int y) {
  if (key == 'q') {
    glutLeaveMainLoop();
    return;
  }
  if (key == 'h') { look_at_2 -= PI / 90; return; }
  if (key == 'j') { look_at_1 -= PI / 90; return; }
  if (key == 'k') { look_at_1 += PI / 90; return; }
  if (key == 'l') { look_at_2 += PI / 90; return; }
  if (key == 'f') { look_at_r /= 0.9f; return; }
  if (key == 'n') { look_at_r *= 0.9f; return; }
}

__global__ void picture(vvv *pos) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= (XX*YY)) return;
  int hh  = index / XX;
  int ww  = index % XX;
  int h = hh - (YY/2);
  int w = ww - (XX/2);
  pos[index].x =  0.0f;
  pos[index].y =  0.001*w;
  pos[index].z = -0.001*h;
  pos[index].r = (float)FRM[PADR(hh, ww, 0)]/255;
  pos[index].g = (float)FRM[PADR(hh, ww, 1)]/255;
  pos[index].b = (float)FRM[PADR(hh, ww, 2)]/255;
}

void set_frm(cv::Mat frame) {
  for (int hh = 0; hh < YY; hh++) {
    for (int ww = 0; ww < XX; ww++) {
      cv::Vec3b bgr = frame.at<cv::Vec3b>(hh, ww);
      h_FRM[PADR(hh, ww, 0)] = bgr[2];
      h_FRM[PADR(hh, ww, 1)] = bgr[1];
      h_FRM[PADR(hh, ww, 2)] = bgr[0];
    }
  }
}

void idle(void) {
  *cap >> frame;
  /*------------------------------*/
  set_frm(frame);
  hipMemcpy(d_FRM, h_FRM, FRM_SIZE, hipMemcpyHostToDevice);
  hipGraphicsMapResources(1, &v_res, NULL);
  vvv *pos;
  hipGraphicsResourceGetMappedPointer((void**)&pos, NULL, v_res);
  picture<<< ((XX*YY)/TD_NUM)+1, TD_NUM >>>(pos);
  hipGraphicsUnmapResources(1, &v_res, NULL);
  /*------------------------------*/
  glutPostRedisplay();
}

/********************************************************************/
int main(int argc, char** argv) {
  look_at_r = LOOK_AT_R;
  look_at_1 = LOOK_AT_1;
  look_at_2 = LOOK_AT_2;
  /*------------------------------*/
  cap = new cv::VideoCapture;
  cap->open(0);
  if (cap->isOpened()) printf("Camera Successfully Opened\n");
  else               { printf("Cannot Open Camera\n"); return 1; }
  /*------------------------------*/
  int gpuCount = 0;
  hipGetDeviceCount(&gpuCount);
  if (gpuCount == 0) { printf("There are no GPU.\n"); return 1; }
  hipSetDevice(0);
  hipDeviceProp_t gpuProp;
  hipGetDeviceProperties(&gpuProp, 0);
  printf("GPU %s is used.\n", gpuProp.name);
  h_FRM = new int[FRM_NUM];
  hipMalloc((void **)&d_FRM, FRM_SIZE);
  hipMemcpyToSymbol(HIP_SYMBOL(FRM), &d_FRM, sizeof(int*));
  cudaGLSetGLDevice(0);
  /*------------------------------*/
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_DEPTH);
  glutInitWindowSize (2*XX, 2*YY);
  glutInitWindowPosition (390, 44);
  glutCreateWindow("K. OGURI 20180130");
  glutDisplayFunc(display);
  glutReshapeFunc(reshape);
  glutKeyboardFunc(keyboard);
  glutIdleFunc(idle);
  /*------------------------------*/
  glewInit();
  GLuint v_buf;
  glGenBuffers(1, &v_buf);
  glBindBuffer(GL_ARRAY_BUFFER, v_buf);
  glBufferData(GL_ARRAY_BUFFER, (XX*YY)*sizeof(vvv), NULL, GL_DYNAMIC_DRAW);
  hipGraphicsGLRegisterBuffer(&v_res, v_buf, cudaGraphicsMapFlagsNone);
  /*------------------------------*/
  glClearColor(0.0, 0.0, 0.0, 1.0);
  glClearDepth(1.0);
  glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_GLUTMAINLOOP_RETURNS);
  glutMainLoop();
  /*------------------------------*/
  hipGraphicsUnregisterResource(v_res);
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  glDeleteBuffers(1, &v_buf);
  hipDeviceReset();
  /*------------------------------*/
  frame.release();
  cap->release();
  delete cap;
  hipFree(d_FRM);
  delete [] h_FRM;
  return 0;
}
